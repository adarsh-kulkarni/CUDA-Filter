#include "hip/hip_runtime.h"
// 
// Filters
//

// Includes: system
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <fcntl.h>
#include <stdint.h>
#include <errno.h>
#include <assert.h>
#include <string.h>
#include <sys/io.h>


#include <cutil_inline.h>

// Includes: local
#include "bmp.h"

enum {SOBEL_FILTER=1, SOBEL_FILTER5x5, AVERAGE_FILTER, HIGH_BOOST_FILTER};

#define CLAMP_8bit(x) max(0, min(255, (x)))

char *BMPInFile = "lena.bmp";
char *BMPOutFile = "output.bmp";
char *Filter = "sobel";
int FilterMode  = SOBEL_FILTER;



float SobelMatrix[25] = {-1,-4,-6,-4,-1,-2,-8,-12,-8,-2,0,0,0,0,0,2,8,12,8,2,1,4,6,4,1};


// Functions
void Cleanup(void);
void ParseArguments(int, char**);
void FilterWrapper(unsigned char* pImageIn, float *sobelMatrix, int Width, int Height);

// Kernels
__global__ void SobelFilter(unsigned char *g_DataIn, unsigned char *g_DataOut, int width, int height);
__global__ void AverageFilter(unsigned char *g_DataIn, unsigned char *g_DataOut, int width, int height);
__global__ void HighBoostFilter(unsigned char *g_DataIn, unsigned char *g_DataOut, int width, int height);
__global__ void SobelFilter5x5(unsigned char *g_DataIn, unsigned char *g_DataOut, float *sobelMatrix, int width, int height);


/* Device Memory */
unsigned char *d_In;
unsigned char *d_Out;
float *d_Sobel;

// Setup for kernel size
const int TILE_WIDTH    = 6;
const int TILE_HEIGHT   = 6;

//const int FILTER_RADIUS = 1;
const int FILTER_RADIUS = 2;

const int FILTER_DIAMETER = 2 * FILTER_RADIUS + 1;
const int FILTER_AREA   = FILTER_DIAMETER * FILTER_DIAMETER;

const int BLOCK_WIDTH   = TILE_WIDTH + 2*FILTER_RADIUS;
const int BLOCK_HEIGHT  = TILE_HEIGHT + 2*FILTER_RADIUS;

const int EDGE_VALUE_THRESHOLD = 40;
const int HIGH_BOOST_FACTOR = 10;


//Timer variables
unsigned int timer_GPU = 0;
unsigned int timer_CPU = 0;


#include "filter_kernel5x5.cu"

void BitMapRead(char *file, struct bmp_header *bmp, struct dib_header *dib, unsigned char **data, unsigned char **palete)
{
   size_t palete_size;
   int fd;

   if((fd = open(file, O_RDONLY )) < 0)
           FATAL("Open Source");

   if(read(fd, bmp, BMP_SIZE) != BMP_SIZE)
           FATAL("Read BMP Header");

   if(read(fd, dib, DIB_SIZE) != DIB_SIZE)
           FATAL("Read DIB Header");

   assert(dib->bpp == 8);

   palete_size = bmp->offset - BMP_SIZE - DIB_SIZE;
   if(palete_size > 0) {
           *palete = (unsigned char *)malloc(palete_size);
           int go = read(fd, *palete, palete_size);
           if (go != palete_size) {
                   FATAL("Read Palete");
           }
   }

   *data = (unsigned char *)malloc(dib->image_size);
   if(read(fd, *data, dib->image_size) != dib->image_size)
           FATAL("Read Image");

   close(fd);
}


void BitMapWrite(char *file, struct bmp_header *bmp, struct dib_header *dib, unsigned char *data, unsigned char *palete)
{
   size_t palete_size;
   int fd;

   palete_size = bmp->offset - BMP_SIZE - DIB_SIZE;

   if((fd = open(file, O_WRONLY | O_CREAT | O_TRUNC,
                             S_IRUSR | S_IWUSR |S_IRGRP)) < 0)
           FATAL("Open Destination");

   if(write(fd, bmp, BMP_SIZE) != BMP_SIZE)
           FATAL("Write BMP Header");

   if(write(fd, dib, DIB_SIZE) != DIB_SIZE)
           FATAL("Write BMP Header");

   if(palete_size != 0) {
           if(write(fd, palete, palete_size) != palete_size)
                   FATAL("Write Palete");
   }
   if(write(fd, data, dib->image_size) != dib->image_size)
           FATAL("Write Image");
   close(fd);
}



void CPU_Sobel(unsigned char* imageIn, unsigned char* imageOut, int width, int height, const float *SobelMatrix)
{
  int i, j, rows, cols, startCol, endCol, startRow, endRow;


  rows = height;
  cols = width;
 
  // Initialize all output pixels to zero 
  for(i=0; i<rows; i++) {
    for(j=0; j<cols; j++) {
	imageOut[i*width + j] = 0;
    }
  }

  startCol = 2;
  endCol = cols - 2;
  startRow = 2;
  endRow = rows - 2;
  
  // Go through all inner pizel positions 
  for(i=startRow; i<endRow; i++) {
    for(j=startCol; j<endCol; j++) {

       // sum up the 9 values to calculate both the direction x and direction y
       float sumX = 0, sumY=0;
       for(int dy = -FILTER_RADIUS; dy <= FILTER_RADIUS; dy++) {
          for(int dx = -FILTER_RADIUS; dx <= FILTER_RADIUS; dx++) {
             float Pixel = (float)(imageIn[i*width + j +  (dy * width + dx)]);
	    
             sumX += Pixel * SobelMatrix[(dy + FILTER_RADIUS) * FILTER_DIAMETER + (dx+FILTER_RADIUS)];
             sumY += Pixel * SobelMatrix[(dx + FILTER_RADIUS) * FILTER_DIAMETER + (dy+FILTER_RADIUS)];
            
	     
          }
	}
       imageOut[i*width + j] = (float)(0.045454545 * (abs(sumX) + abs(sumY))) > EDGE_VALUE_THRESHOLD ? 255 : 0;
    }
  }
}




// Host code
int main(int argc, char** argv)
{
    ParseArguments(argc, argv);

    struct bmp_header bmp;
    struct dib_header dib;

    unsigned char *palete = NULL;
    unsigned char *data = NULL, *out = NULL;
  


    cutilCheckError(cutCreateTimer(&timer_CPU)); 
    cutilCheckError(cutCreateTimer(&timer_GPU));

    printf("Running %s filter\n", Filter);
    BitMapRead(BMPInFile, &bmp, &dib, &data, &palete);
    out = (unsigned char *)malloc(dib.image_size);

    printf("Computing the CPU output\n");
    printf("Image details: %d by %d = %d , imagesize = %d\n", dib.width, dib.height, dib.width * dib.height,dib.image_size);

   
    cutilCheckError(cutStartTimer(timer_CPU));
    
    CPU_Sobel(data, out, dib.width, dib.height, SobelMatrix);
   
    cutilCheckError(cutStopTimer(timer_CPU));
    
   
    BitMapWrite("CPU_sobel.bmp", &bmp, &dib, out, palete);
    printf("Done with CPU output\n");

    //GPU timer code

    printf("Allocating %d bytes for image \n", dib.image_size);
    cutilSafeCall( hipMalloc( (void **)&d_In, dib.image_size*sizeof(unsigned char)) );
    cutilSafeCall( hipMalloc( (void **)&d_Out, dib.image_size*sizeof(unsigned char)) );
    cutilSafeCall( hipMalloc( (void **)&d_Sobel, 25*sizeof(float)) );

    cutilCheckError(cutStartTimer(timer_GPU)); 
    hipMemcpy(d_In, data, dib.image_size*sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_Sobel, &SobelMatrix, 25*sizeof(float), hipMemcpyHostToDevice);  
 
    FilterWrapper(data, d_Sobel, dib.width, dib.height);

    // Copy image back to host
    hipMemcpy(out, d_Out, dib.image_size*sizeof(unsigned char), hipMemcpyDeviceToHost);

    cutilCheckError(cutStopTimer(timer_GPU));	 

    // Write output image   
    BitMapWrite(BMPOutFile, &bmp, &dib, out, palete);

    printf("GPU Execution time : %f (ms) \n",cutGetTimerValue(timer_GPU));	
    printf("CPU Execution time : %f (ms) \n",cutGetTimerValue(timer_CPU));

    Cleanup();
}

void Cleanup(void)
{
    //Destroy timer value
    cutilCheckError(cutDeleteTimer(timer_GPU));
    cutilCheckError(cutDeleteTimer(timer_CPU));


    cutilSafeCall( hipDeviceReset() );
    exit(0);
}


void FilterWrapper(unsigned char* pImageIn, float *SobelMatrix, int Width, int Height)
{
   // Design grid disection around tile size
   int gridWidth  = (Width + TILE_WIDTH - 1) / TILE_WIDTH;
   int gridHeight = (Height + TILE_HEIGHT - 1) / TILE_HEIGHT;
   dim3 dimGrid(gridWidth, gridHeight);

   // But actually invoke larger blocks to take care of surrounding shared memory
   dim3 dimBlock(BLOCK_WIDTH, BLOCK_HEIGHT);

   switch(FilterMode) {
     case SOBEL_FILTER:
     printf("Sobel Filter \n");
     SobelFilter<<< dimGrid, dimBlock >>>(d_In, d_Out, Width, Height);
   
     cutilCheckMsg("kernel launch failure");
     break;
     case SOBEL_FILTER5x5:
     printf("Sobel Filter 5x5 \n");
     SobelFilter5x5<<< dimGrid, dimBlock >>>(d_In, d_Out, d_Sobel, Width, Height);
     cutilCheckMsg("kernel launch failure");
     break;
     case AVERAGE_FILTER:
     printf("Average Filter \n");
     AverageFilter<<< dimGrid, dimBlock >>>(d_In, d_Out, Width, Height);
     cutilCheckMsg("kernel launch failure");
     break;
     case HIGH_BOOST_FILTER:
     printf("Boost Filter \n");
     HighBoostFilter<<< dimGrid, dimBlock >>>(d_In, d_Out, Width, Height);
     cutilCheckMsg("kernel launch failure");
     break;
    }
   cutilSafeCall( hipDeviceSynchronize() );
}



// Parse program arguments
void ParseArguments(int argc, char** argv)
{
    for (int i = 0; i < argc; ++i) {
        if (strcmp(argv[i], "--file") == 0 || strcmp(argv[i], "-file") == 0) {
            BMPInFile = argv[i+1];
	    i = i + 1;
        }
        if (strcmp(argv[i], "--out") == 0 || strcmp(argv[i], "-out") == 0) {
            BMPOutFile = argv[i+1];
	    i = i + 1;
        }
        if (strcmp(argv[i], "--filter") == 0 || strcmp(argv[i], "-filter") == 0) {
            Filter = argv[i+1];
	    i = i + 1;
            if (strcmp(Filter, "sobel") == 0)
		FilterMode = SOBEL_FILTER;
   	    else if (strcmp(Filter, "sobel5x5") == 0)
		FilterMode = SOBEL_FILTER5x5;
            else if (strcmp(Filter, "average") == 0)
		FilterMode = AVERAGE_FILTER;
            else if (strcmp(Filter, "boost") == 0)
		FilterMode = HIGH_BOOST_FILTER;
	 
        }
    }
}



